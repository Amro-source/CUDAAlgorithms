#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void vectorAdd(const float *a, const float *b, float *c, int numElements)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	if (i < numElements)
	{
		c[i] = 5*a[i] +3* b[i];
	}
}

int main(int argc, char *argv[])
{
	int numElements = 5e+4;
	// Allocate vectors a, b and c in host memory.
	size_t numBytes = sizeof(float)* numElements;


	float *h_a = (float *)malloc(numBytes);
	float *h_b = (float *)malloc(numBytes);
	float *h_c = (float *)malloc(numBytes);


	// Initialize vectors a and b.
	for (int i = 0; i < numElements; ++i)
	{
		h_a[i] = rand() / (float)RAND_MAX;
		h_b[i] = rand() / (float)RAND_MAX;
	}

	printf("dumping some arrays elements values\n");
	for (int i = 0; i < 10; ++i)
	{
		printf("%d \n", h_a[i]);
		printf("%d \n", h_b[i]);

	}


	printf("End of dumping\n");

	// Allocate vectors a, b and c in device memory.
	float *d_a;
	float *d_b;
	float *d_c;

	printf("Parallelizing a big task for GPU card\n");


	hipMalloc((void **)&d_a, numBytes);
	hipMalloc((void **)&d_b, numBytes);
	hipMalloc((void **)&d_c, numBytes);
	// Copy vectors a and b from host memory to device memory synchronously.
	hipMemcpy(d_a, h_a, numBytes, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, numBytes, hipMemcpyHostToDevice);


	// Determine the number of threads per block and the number of blocks per grid.
	int numThreadsPerBlock = 256;
	int numBlocksPerGrid = (numElements + numThreadsPerBlock - 1) / numThreadsPerBlock;
	// Invoke the kernel on device asynchronously.


	vectorAdd << <numBlocksPerGrid, numThreadsPerBlock >> >(d_a, d_b, d_c, numElements);
	// Copy vector c from device memory to host memory synchronously.
	hipMemcpy(h_c, d_c, numBytes, hipMemcpyDeviceToHost);
	// Validate the result.

	for (int i = 0; i < numElements; ++i)
	{
		float actual = h_c[i];
		float expected = h_a[i] + h_b[i];

		printf("h_a[%d]=%f  \n",i,h_a[i]);
		printf("h_b[%d]=%f \n",  i, h_b[i]);

		printf("h_c[%d]=%f\n",i, h_c[i]);


		/*if (fabs(actual - expected) > 1e-7)
		{
			printf("h_c[%d] = %f, expected = %f\n", i, actual, expected);
			break;
		}*/
	}
	// Cleanup.
	hipFree(d_c);
	hipFree(d_b);
	hipFree(d_a);
	hipDeviceReset();
	/*free(h_c);
	free(h_b);
	free(h_a);*/

}